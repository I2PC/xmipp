#include "hip/hip_runtime.h"
#ifndef CUDA_FORWARD_ART_ZERNIKE3D_CU
#define CUDA_FORWARD_ART_ZERNIKE3D_CU

// Compilation settings

#ifndef KTT_USED
#include "cuda_forward_art_zernike3d_defines.h"
#include "cuda_forward_art_zernike3d.h"
#endif

#if USE_DOUBLE_PRECISION == 1
// Types
using PrecisionType = double;
using PrecisionType3 = double3;
// Constants
#define _PI_ (3.1415926535897931e+0)
// Functions
#define SQRT sqrt
#define ATAN2 atan2
#define COS cos
#define SIN sin
#define CUDA_FLOOR floor

#else
// Types
using PrecisionType = float;
using PrecisionType3 = float3;
// Constants
#define _PI_ (3.1415926535897f)
// Functions
#define SQRT sqrtf
#define ATAN2 atan2f
#define COS cosf
#define SIN sinf
#define CUDA_FLOOR floorf

#endif// USE_DOUBLE_PRECISION

__device__ __inline__ PrecisionType shfl_down(PrecisionType val, int offset) {
#if (CUDART_VERSION >= 9000)
    return __shfl_down_sync(0xFFFFFFFF, val, offset);
#else
    return __shfl_down(val, offset);
#endif
}

// Compilation settings - end

// Define data structures
#ifdef KTT_USED

struct ImageMetaData
{
    int xShift;
    int yShift;
    int zShift;

    int xDim;
    int yDim;
    int zDim;
}

struct Volumes
{
    PrecisionType* I;
    PrecisionType* R;
    unsigned count;
    unsigned volumeSize;
};

struct IROimages
{
    PrecisionType* VI;
    PrecisionType* VR;
    PrecisionType* VO;
};

struct DeformImages
{
    PrecisionType* Gx;
    PrecisionType* Gy;
    PrecisionType* Gz;
};
#endif// KTT_USED

// CUDA kernel defines
#define BLOCK_SIZE (BLOCK_X_DIM * BLOCK_Y_DIM * BLOCK_Z_DIM)

// ImageData macros

// Index to global memory
#define GET_IDX(ImD,k,i,j) \
    ((ImD).xDim * (ImD).yDim * (k) + (ImD).xDim * (i) + (j))

// Logical index = Physical index + shift
#define P2L_X_IDX(ImD,j) \
    ((j) + (ImD).xShift)

#define P2L_Y_IDX(ImD,i) \
    ((i) + (ImD).yShift)

#define P2L_Z_IDX(ImD,k) \
    ((k) + (ImD).zShift)

// Physical index = Logical index - shift
#define L2P_X_IDX(ImD,j) \
    ((j) - (ImD).xShift)

#define L2P_Y_IDX(ImD,i) \
    ((i) - (ImD).yShift)

#define L2P_Z_IDX(ImD,k) \
    ((k) - (ImD).zShift)

// Element access
#define ELEM_3D(ImD,meta,k,i,j) \
    ((ImD)[GET_IDX((meta), (k), (i), (j))])

#define ELEM_3D_SHIFTED(ImD,meta,k,i,j) \
    (ELEM_3D((ImD), (meta), (k) - (meta).zShift, (i) - (meta).yShift, (j) - (meta).xShift))

// Utility macros
#define IS_OUTSIDE(ImD,k,i,j) \
    ((j) < (ImD).xShift || (j) > (ImD).xShift + (ImD).xDim - 1 || \
     (i) < (ImD).yShift || (i) > (ImD).yShift + (ImD).yDim - 1 || \
     (k) < (ImD).zShift || (k) > (ImD).zShift + (ImD).zDim - 1)

#define IS_OUTSIDE_PHYS(ImD,k,i,j) \
    ((j) < 0 || (ImD).xDim <= (j) || \
     (i) < 0 || (ImD).yDim <= (i) || \
     (k) < 0 || (ImD).zDim <= (k))

// Smart casting to selected precision (at compile time)
// ...just shorter static_cast
#define CST(num) (static_cast<PrecisionType>((num)))

#define FLOOR(x) (((x) == (int)(x)) ? (int)(x):(((x) > 0) ? (int)(x) : \
                  (int)((x) - 1)))
#define LIN_INTERP(a, l, h) ((l) + ((h) - (l)) * (a))

// Forward declarations
__device__ PrecisionType ZernikeSphericalHarmonics(int l1, int n, int l2, int m,
        PrecisionType xr, PrecisionType yr, PrecisionType zr, PrecisionType r);

__device__ PrecisionType interpolatedElement3D(
        PrecisionType* ImD, ImageMetaData imgMeta,
        PrecisionType x, PrecisionType y, PrecisionType z,
        PrecisionType doutside_value = 0);

/*
 * The beast
 */
extern "C" __global__ void computeDeform(
        PrecisionType Rmax2,
        PrecisionType iRmax,
        IROimages images,
        int4* zshparams,
        PrecisionType3* clnm,
        unsigned steps,
        ImageMetaData imageMetaData,
        Volumes volumes,
        DeformImages deformImages,
        bool applyTransformation,
        bool saveDeformation,
        PrecisionType* outArrayGlobal
        )
{
    extern __shared__ char sharedBuffer[];
    unsigned sharedBufferOffset = 0;

    // Thread index in a block
    unsigned tIdx = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    // Get physical indexes
    int kPhys = blockIdx.z * blockDim.z + threadIdx.z;
    int iPhys = blockIdx.y * blockDim.y + threadIdx.y;
    int jPhys = blockIdx.x * blockDim.x + threadIdx.x;

    // Update to logical indexes (calculations expect logical indexing)
    int k = P2L_Z_IDX(imageMetaData, kPhys);
    int i = P2L_Y_IDX(imageMetaData, iPhys);
    int j = P2L_X_IDX(imageMetaData, jPhys);


    int4* zshShared = (int4*)(sharedBuffer + sharedBufferOffset);
    sharedBufferOffset += sizeof(int4) * steps;

    PrecisionType3* clnmShared = (PrecisionType3*)(sharedBuffer + sharedBufferOffset);
    sharedBufferOffset += sizeof(PrecisionType3) * steps;

    // Load zsh, clnm parameters to the shared memory
    if (steps <= BLOCK_SIZE) {
        if (tIdx < steps) {
            zshShared[tIdx] = zshparams[tIdx];
            clnmShared[tIdx] = clnm[tIdx];
        }
    } else {
        if (tIdx == 0) {
            for (unsigned idx = 0; idx < steps; idx++) {
                zshShared[idx] = zshparams[idx];
                clnmShared[idx] = clnm[idx];
            }
        }
    }

    __syncthreads();

    // Define and compute necessary values
    PrecisionType r2 = k*k + i*i + j*j;
    PrecisionType rr = SQRT(r2) * iRmax;
    PrecisionType gx = 0.0, gy = 0.0, gz = 0.0;

    if (r2 < Rmax2) {
        for (int idx = 0; idx < steps; idx++) {
            int l1 = zshShared[idx].w;
            int n = zshShared[idx].x;
            int l2 = zshShared[idx].y;
            int m = zshShared[idx].z;

            PrecisionType zsph = ZernikeSphericalHarmonics(l1, n, l2, m,
                    j * iRmax, i * iRmax, k * iRmax, rr);

            if (rr > 0 || l2 == 0) {
                gx += zsph * clnmShared[idx].x;
                gy += zsph * clnmShared[idx].y;
                gz += zsph * clnmShared[idx].z;
            }
        }
    }

    PrecisionType voxelI, voxelR;
    PrecisionType diff;

    PrecisionType localDiff2 = 0.0, localSumVD = 0.0, localModg = 0.0;

    bool isOutside = IS_OUTSIDE_PHYS(imageMetaData, kPhys, iPhys, jPhys);

    if (applyTransformation && !isOutside) {
        // Logical indexes used to check whether the point is in the matrix
        voxelI = interpolatedElement3D(images.VI, imageMetaData,
                j + gx, i + gy, k + gz);

        ELEM_3D(images.VO, imageMetaData, kPhys, iPhys, jPhys) = voxelI;
    }

    if (!isOutside) {
        for (unsigned idv = 0; idv < volumes.count; idv++) {
            voxelR = ELEM_3D(volumes.R + idv * volumes.volumeSize,
                    imageMetaData, kPhys, iPhys, jPhys);
            voxelI = interpolatedElement3D(volumes.I + idv * volumes.volumeSize,
                    imageMetaData, j + gx, i + gy, k + gz);

            if (voxelI >= 0.0)
                localSumVD += voxelI;

            diff = voxelR - voxelI;
            localDiff2 += diff * diff;
        }
        localModg += volumes.count * (gx*gx + gy*gy + gz*gz);
    }

    __shared__ PrecisionType diff2Shared[BLOCK_SIZE];
    __shared__ PrecisionType sumVDShared[BLOCK_SIZE];
    __shared__ PrecisionType modfgShared[BLOCK_SIZE];

    diff2Shared[tIdx] = localDiff2;
    sumVDShared[tIdx] = localSumVD;
    modfgShared[tIdx] = localModg;

    __syncthreads();

    // First level of conditions are evaluated during compilation
    if (BLOCK_SIZE >= 1024) {
        if (tIdx < 512) {
            diff2Shared[tIdx] += diff2Shared[tIdx + 512];
            sumVDShared[tIdx] += sumVDShared[tIdx + 512];
            modfgShared[tIdx] += modfgShared[tIdx + 512];
        }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 512) {
        if (tIdx < 256) {
            diff2Shared[tIdx] += diff2Shared[tIdx + 256];
            sumVDShared[tIdx] += sumVDShared[tIdx + 256];
            modfgShared[tIdx] += modfgShared[tIdx + 256];
        }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 256) {
        if (tIdx < 128) {
            diff2Shared[tIdx] += diff2Shared[tIdx + 128];
            sumVDShared[tIdx] += sumVDShared[tIdx + 128];
            modfgShared[tIdx] += modfgShared[tIdx + 128];
        }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 128) {
        if (tIdx < 64) {
            diff2Shared[tIdx] += diff2Shared[tIdx + 64];
            sumVDShared[tIdx] += sumVDShared[tIdx + 64];
            modfgShared[tIdx] += modfgShared[tIdx + 64];
        }
        __syncthreads();
    }
    // Last warp reduction
    if (tIdx < 32) {
        localDiff2 = diff2Shared[tIdx];
        localSumVD = sumVDShared[tIdx];
        localModg = modfgShared[tIdx];
        if (BLOCK_SIZE >= 64) {
            localDiff2 += diff2Shared[tIdx + 32];
            localSumVD += sumVDShared[tIdx + 32];
            localModg += modfgShared[tIdx + 32];
        }
        // Reduce warp
        for (int offset = 32 / 2; offset > 0; offset >>= 1) {
            localDiff2 += shfl_down(localDiff2, offset);
            localSumVD += shfl_down(localSumVD, offset);
            localModg += shfl_down(localModg, offset);
        }
    }

    // Save values to the global memory for later
    if (tIdx == 0) {
        unsigned bIdx = blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x;
        unsigned GRID_SIZE = gridDim.x * gridDim.y * gridDim.z;
        // Resulting values are in variables local* => no need to go into shared mem
        outArrayGlobal[bIdx] = localDiff2;
        outArrayGlobal[bIdx + GRID_SIZE] = localSumVD;
        outArrayGlobal[bIdx + GRID_SIZE * 2] = localModg;
    }

    if (saveDeformation && !isOutside) {
        ELEM_3D(deformImages.Gx, imageMetaData, kPhys, iPhys, jPhys) = gx;
        ELEM_3D(deformImages.Gy, imageMetaData, kPhys, iPhys, jPhys) = gy;
        ELEM_3D(deformImages.Gz, imageMetaData, kPhys, iPhys, jPhys) = gz;
    }
}

/*
 * Linear interpolation
 */
__device__ PrecisionType interpolatedElement3D(
        PrecisionType* ImD, ImageMetaData imgMeta,
        PrecisionType x, PrecisionType y, PrecisionType z,
        PrecisionType outside_value)
{
        int x0 = (int)CUDA_FLOOR(x);
        PrecisionType fx = x - x0;
        int x1 = x0 + 1;

        int y0 = (int)CUDA_FLOOR(y);
        PrecisionType fy = y - y0;
        int y1 = y0 + 1;

        int z0 = (int)CUDA_FLOOR(z);
        PrecisionType fz = z - z0;
        int z1 = z0 + 1;

        PrecisionType d000 = (IS_OUTSIDE(imgMeta, z0, y0, x0)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, imgMeta, z0, y0, x0);
        PrecisionType d001 = (IS_OUTSIDE(imgMeta, z0, y0, x1)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, imgMeta, z0, y0, x1);
        PrecisionType d010 = (IS_OUTSIDE(imgMeta, z0, y1, x0)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, imgMeta, z0, y1, x0);
        PrecisionType d011 = (IS_OUTSIDE(imgMeta, z0, y1, x1)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, imgMeta, z0, y1, x1);
        PrecisionType d100 = (IS_OUTSIDE(imgMeta, z1, y0, x0)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, imgMeta, z1, y0, x0);
        PrecisionType d101 = (IS_OUTSIDE(imgMeta, z1, y0, x1)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, imgMeta, z1, y0, x1);
        PrecisionType d110 = (IS_OUTSIDE(imgMeta, z1, y1, x0)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, imgMeta, z1, y1, x0);
        PrecisionType d111 = (IS_OUTSIDE(imgMeta, z1, y1, x1)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, imgMeta, z1, y1, x1);

        PrecisionType dx00 = LIN_INTERP(fx, d000, d001);
        PrecisionType dx01 = LIN_INTERP(fx, d100, d101);
        PrecisionType dx10 = LIN_INTERP(fx, d010, d011);
        PrecisionType dx11 = LIN_INTERP(fx, d110, d111);
        PrecisionType dxy0 = LIN_INTERP(fy, dx00, dx10);
        PrecisionType dxy1 = LIN_INTERP(fy, dx01, dx11);

        return LIN_INTERP(fz, dxy0, dxy1);
}

/*
 * ZSH
 */
__forceinline__ __device__ PrecisionType ZernikeSphericalHarmonics(int l1, int n, int l2, int m, PrecisionType xr, PrecisionType yr, PrecisionType zr, PrecisionType rr)
{
            // General variables
            PrecisionType r2 = rr * rr, xr2 = xr * xr, yr2 = yr * yr,
                          zr2 = zr * zr;

#if L2 >= 5
            // Variables needed for l2 >= 5
            PrecisionType tht = CST(0.0), phi = CST(0.0), cost = CST(0.0),
                          sint = CST(0.0), cost2 = CST(0.0), sint2 = CST(0.0);
            if (l2 >= 5) {
              tht = ATAN2(yr, xr);
              phi = ATAN2(zr, SQRT(xr2 + yr2));
              sint = SIN(phi);
              cost = COS(tht);
              sint2 = sint * sint;
              cost2 = cost * cost;
            }
#endif// L2 >= 5

            // Zernike polynomial
            PrecisionType R = CST(0.0);

            switch (l1) {
            case 0:
              R = SQRT(CST(3));
              break;
            case 1:
              R = SQRT(CST(5)) * rr;
              break;
            case 2:
              switch (n) {
              case 0:
                R = CST(-0.5) * SQRT(CST(7)) *
                    (CST(2.5) * (1 - 2 * r2) + CST(0.5));
                break;
              case 2:
                R = SQRT(CST(7)) * r2;
                break;
              }
              break;
#if L1 >= 3
            case 3:
              switch (n) {
              case 1:
                R = CST(-1.5) * rr * (CST(3.5) * (1 - 2 * r2) + CST(1.5));
                break;
              case 3:
                R = 3 * r2 * rr;
              }
              break;
#endif// L1 >= 3
#if L1 >= 4
            case 4:
              switch (n) {
              case 0:
                R = SQRT(CST(11)) *
                    ((63 * r2 * r2 / 8) - (35 * r2 / 4) + (CST(15) / CST(8)));
                break;
              case 2:
                R = CST(-0.5) * SQRT(CST(11)) * r2 *
                    (CST(4.5) * (1 - 2 * r2) + CST(2.5));
                break;
              case 4:
                R = SQRT(CST(11)) * r2 * r2;
                break;
              }
              break;
#endif// L1 >= 4
#if L1 >= 5
            case 5:
              switch (n) {
              case 1:
                R = SQRT(CST(13)) * rr *
                    ((99 * r2 * r2 / 8) - (63 * r2 / 4) + (CST(35) / CST(8)));
                break;
              case 3:
                R = CST(-0.5) * SQRT(CST(13)) * r2 * rr *
                    (CST(5.5) * (1 - 2 * r2) + CST(3.5));
                break;
              }
              break;
#endif// L1 >= 5
            }

            // Spherical harmonic
            PrecisionType Y = CST(0.0);

            switch (l2) {
            case 0:
              Y = (CST(1.0) / CST(2.0)) * SQRT((PrecisionType)CST(1.0) / _PI_);
              break;
            case 1:
              switch (m) {
              case -1:
                Y = SQRT(CST(3.0) / (CST(4.0) * _PI_)) * yr;
                break;
              case 0:
                Y = SQRT(CST(3.0) / (CST(4.0) * _PI_)) * zr;
                break;
              case 1:
                Y = SQRT(CST(3.0) / (CST(4.0) * _PI_)) * xr;
                break;
              }
              break;
            case 2:
              switch (m) {
              case -2:
                Y = SQRT(CST(15.0) / (CST(4.0) * _PI_)) * xr * yr;
                break;
              case -1:
                Y = SQRT(CST(15.0) / (CST(4.0) * _PI_)) * zr * yr;
                break;
              case 0:
                Y = SQRT(CST(5.0) / (CST(16.0) * _PI_)) *
                    (-xr2 - yr2 + CST(2.0) * zr2);
                break;
              case 1:
                Y = SQRT(CST(15.0) / (CST(4.0) * _PI_)) * xr * zr;
                break;
              case 2:
                Y = SQRT(CST(15.0) / (CST(16.0) * _PI_)) * (xr2 - yr2);
                break;
              }
              break;
#if L2 >= 3
            case 3:
              switch (m) {
              case -3:
                Y = SQRT(CST(35.0) / (CST(16.0) * CST(2.0) * _PI_)) * yr *
                    (CST(3.0) * xr2 - yr2);
                break;
              case -2:
                Y = SQRT(CST(105.0) / (CST(4.0) * _PI_)) * zr * yr * xr;
                break;
              case -1:
                Y = SQRT(CST(21.0) / (CST(16.0) * CST(2.0) * _PI_)) * yr *
                    (CST(4.0) * zr2 - xr2 - yr2);
                break;
              case 0:
                Y = SQRT(CST(7.0) / (CST(16.0) * _PI_)) * zr *
                    (CST(2.0) * zr2 - CST(3.0) * xr2 - CST(3.0) * yr2);
                break;
              case 1:
                Y = SQRT(CST(21.0) / (CST(16.0) * CST(2.0) * _PI_)) * xr *
                    (CST(4.0) * zr2 - xr2 - yr2);
                break;
              case 2:
                Y = SQRT(CST(105.0) / (CST(16.0) * _PI_)) * zr * (xr2 - yr2);
                break;
              case 3:
                Y = SQRT(CST(35.0) / (CST(16.0) * CST(2.0) * _PI_)) * xr *
                    (xr2 - CST(3.0) * yr2);
                break;
              }
              break;
#endif// L2 >= 3
#if L2 >= 4
            case 4:
              switch (m) {
              case -4:
                Y = SQRT((CST(35.0) * CST(9.0)) / (CST(16.0) * _PI_)) * yr *
                    xr * (xr2 - yr2);
                break;
              case -3:
                Y = SQRT((CST(9.0) * CST(35.0)) /
                         (CST(16.0) * CST(2.0) * _PI_)) *
                    yr * zr * (CST(3.0) * xr2 - yr2);
                break;
              case -2:
                Y = SQRT((CST(9.0) * CST(5.0)) / (CST(16.0) * _PI_)) * yr * xr *
                    (CST(7.0) * zr2 - (xr2 + yr2 + zr2));
                break;
              case -1:
                Y = SQRT((CST(9.0) * CST(5.0)) /
                         (CST(16.0) * CST(2.0) * _PI_)) *
                    yr * zr * (CST(7.0) * zr2 - CST(3.0) * (xr2 + yr2 + zr2));
                break;
              case 0:
                Y = SQRT(CST(9.0) / (CST(16.0) * CST(16.0) * _PI_)) *
                    (CST(35.0) * zr2 * zr2 - CST(30.0) * zr2 + CST(3.0));
                break;
              case 1:
                Y = SQRT((CST(9.0) * CST(5.0)) /
                         (CST(16.0) * CST(2.0) * _PI_)) *
                    xr * zr * (CST(7.0) * zr2 - CST(3.0) * (xr2 + yr2 + zr2));
                break;
              case 2:
                Y = SQRT((CST(9.0) * CST(5.0)) / (CST(8.0) * CST(8.0) * _PI_)) *
                    (xr2 - yr2) * (CST(7.0) * zr2 - (xr2 + yr2 + zr2));
                break;
              case 3:
                Y = SQRT((CST(9.0) * CST(35.0)) /
                         (CST(16.0) * CST(2.0) * _PI_)) *
                    xr * zr * (xr2 - CST(3.0) * yr2);
                break;
              case 4:
                Y = SQRT((CST(9.0) * CST(35.0)) /
                         (CST(16.0) * CST(16.0) * _PI_)) *
                    (xr2 * (xr2 - CST(3.0) * yr2) -
                     yr2 * (CST(3.0) * xr2 - yr2));
                break;
              }
              break;
#endif// L2 >= 4
#if L2 >= 5
            case 5:
              switch (m) {
              case -5:
                Y = (CST(3.0) / CST(16.0)) *
                    SQRT(CST(77.0) / (CST(2.0) * _PI_)) * sint2 * sint2 * sint *
                    SIN(CST(5.0) * phi);
                break;
              case -4:
                Y = (CST(3.0) / CST(8.0)) *
                    SQRT(CST(385.0) / (CST(2.0) * _PI_)) * sint2 * sint2 *
                    SIN(CST(4.0) * phi);
                break;
              case -3:
                Y = (CST(1.0) / CST(16.0)) *
                    SQRT(CST(385.0) / (CST(2.0) * _PI_)) * sint2 * sint *
                    (CST(9.0) * cost2 - CST(1.0)) * SIN(CST(3.0) * phi);
                break;
              case -2:
                Y = (CST(1.0) / CST(4.0)) *
                    SQRT(CST(1155.0) / (CST(4.0) * _PI_)) * sint2 *
                    (CST(3.0) * cost2 * cost - cost) * SIN(CST(2.0) * phi);
                break;
              case -1:
                Y = (CST(1.0) / CST(8.0)) *
                    SQRT(CST(165.0) / (CST(4.0) * _PI_)) * sint *
                    (CST(21.0) * cost2 * cost2 - CST(14.0) * cost2 + 1) *
                    SIN(phi);
                break;
              case 0:
                Y = (CST(1.0) / CST(16.0)) * SQRT(CST(11.0) / _PI_) *
                    (CST(63.0) * cost2 * cost2 * cost -
                     CST(70.0) * cost2 * cost + CST(15.0) * cost);
                break;
              case 1:
                Y = (CST(1.0) / CST(8.0)) *
                    SQRT(CST(165.0) / (CST(4.0) * _PI_)) * sint *
                    (CST(21.0) * cost2 * cost2 - CST(14.0) * cost2 + 1) *
                    COS(phi);
                break;
              case 2:
                Y = (CST(1.0) / CST(4.0)) *
                    SQRT(CST(1155.0) / (CST(4.0) * _PI_)) * sint2 *
                    (CST(3.0) * cost2 * cost - cost) * COS(CST(2.0) * phi);
                break;
              case 3:
                Y = (CST(1.0) / CST(16.0)) *
                    SQRT(CST(385.0) / (CST(2.0) * _PI_)) * sint2 * sint *
                    (CST(9.0) * cost2 - CST(1.0)) * COS(CST(3.0) * phi);
                break;
              case 4:
                Y = (CST(3.0) / CST(8.0)) *
                    SQRT(CST(385.0) / (CST(2.0) * _PI_)) * sint2 * sint2 *
                    COS(CST(4.0) * phi);
                break;
              case 5:
                Y = (CST(3.0) / CST(16.0)) *
                    SQRT(CST(77.0) / (CST(2.0) * _PI_)) * sint2 * sint2 * sint *
                    COS(CST(5.0) * phi);
                break;
              }
              break;
#endif// L2 >= 5
            }

            return R * Y;
}


#endif //CUDA_FORWARD_ART_ZERNIKE3D_CU