#include "hip/hip_runtime.h"
#ifndef CUDA_FORWARD_ART_ZERNIKE3D_CU
#define CUDA_FORWARD_ART_ZERNIKE3D_CU

#include "cuda_forward_art_zernike3d.h"
#include "cuda_forward_art_zernike3d_defines.h"

namespace cuda_forward_art_zernike3D {

// Constants
static constexpr float CUDA_PI = 3.1415926535897f;
// Functions
#define SQRT sqrtf
#define ATAN2 atan2f
#define COS cosf
#define SIN sinf
#define CUDA_FLOOR floorf
#define CUDA_ROUND lroundf

#define IS_OUTSIDE2D(ImD, i, j) \
	((j) < STARTINGX((ImD)) || (j) > FINISHINGX((ImD)) || (i) < STARTINGY((ImD)) || (i) > FINISHINGY((ImD)))

// Smart casting to selected precision (at compile time)
// ...just shorter static_cast
#define CST(num) (static_cast<PrecisionType>((num)))

#define LIN_INTERP(a, l, h) ((l) + ((h) - (l)) * (a))

namespace device {

	template<typename PrecisionType>
	__forceinline__ __device__ PrecisionType ZernikeSphericalHarmonics(int l1,
																	   int n,
																	   int l2,
																	   int m,
																	   PrecisionType xr,
																	   PrecisionType yr,
																	   PrecisionType zr,
																	   PrecisionType rr)
	{
		// General variables
		PrecisionType r2 = rr * rr, xr2 = xr * xr, yr2 = yr * yr, zr2 = zr * zr;

#if L2 >= 5
		// Variables needed for l2 >= 5
		PrecisionType tht = CST(0.0), phi = CST(0.0), cost = CST(0.0), sint = CST(0.0), cost2 = CST(0.0),
					  sint2 = CST(0.0);
		if (l2 >= 5) {
			tht = ATAN2(yr, xr);
			phi = ATAN2(zr, SQRT(xr2 + yr2));
			sint = SIN(phi);
			cost = COS(tht);
			sint2 = sint * sint;
			cost2 = cost * cost;
		}
#endif	// L2 >= 5

		// Zernike polynomial
		PrecisionType R = CST(0.0);

		switch (l1) {
			case 0:
				R = SQRT(CST(3));
				break;
			case 1:
				R = SQRT(CST(5)) * rr;
				break;
			case 2:
				switch (n) {
					case 0:
						R = CST(-0.5) * SQRT(CST(7)) * (CST(2.5) * (1 - 2 * r2) + CST(0.5));
						break;
					case 2:
						R = SQRT(CST(7)) * r2;
						break;
				}
				break;
#if L1 >= 3
			case 3:
				switch (n) {
					case 1:
						R = CST(-1.5) * rr * (CST(3.5) * (1 - 2 * r2) + CST(1.5));
						break;
					case 3:
						R = 3 * r2 * rr;
				}
				break;
#endif	// L1 >= 3
#if L1 >= 4
			case 4:
				switch (n) {
					case 0:
						R = SQRT(CST(11)) * ((63 * r2 * r2 / 8) - (35 * r2 / 4) + (CST(15) / CST(8)));
						break;
					case 2:
						R = CST(-0.5) * SQRT(CST(11)) * r2 * (CST(4.5) * (1 - 2 * r2) + CST(2.5));
						break;
					case 4:
						R = SQRT(CST(11)) * r2 * r2;
						break;
				}
				break;
#endif	// L1 >= 4
#if L1 >= 5
			case 5:
				switch (n) {
					case 1:
						R = SQRT(CST(13)) * rr * ((99 * r2 * r2 / 8) - (63 * r2 / 4) + (CST(35) / CST(8)));
						break;
					case 3:
						R = CST(-0.5) * SQRT(CST(13)) * r2 * rr * (CST(5.5) * (1 - 2 * r2) + CST(3.5));
						break;
				}
				break;
#endif	// L1 >= 5
		}

		// Spherical harmonic
		PrecisionType Y = CST(0.0);

		switch (l2) {
			case 0:
				Y = (CST(1.0) / CST(2.0)) * SQRT((PrecisionType)CST(1.0) / CUDA_PI);
				break;
			case 1:
				switch (m) {
					case -1:
						Y = SQRT(CST(3.0) / (CST(4.0) * CUDA_PI)) * yr;
						break;
					case 0:
						Y = SQRT(CST(3.0) / (CST(4.0) * CUDA_PI)) * zr;
						break;
					case 1:
						Y = SQRT(CST(3.0) / (CST(4.0) * CUDA_PI)) * xr;
						break;
				}
				break;
			case 2:
				switch (m) {
					case -2:
						Y = SQRT(CST(15.0) / (CST(4.0) * CUDA_PI)) * xr * yr;
						break;
					case -1:
						Y = SQRT(CST(15.0) / (CST(4.0) * CUDA_PI)) * zr * yr;
						break;
					case 0:
						Y = SQRT(CST(5.0) / (CST(16.0) * CUDA_PI)) * (-xr2 - yr2 + CST(2.0) * zr2);
						break;
					case 1:
						Y = SQRT(CST(15.0) / (CST(4.0) * CUDA_PI)) * xr * zr;
						break;
					case 2:
						Y = SQRT(CST(15.0) / (CST(16.0) * CUDA_PI)) * (xr2 - yr2);
						break;
				}
				break;
#if L2 >= 3
			case 3:
				switch (m) {
					case -3:
						Y = SQRT(CST(35.0) / (CST(16.0) * CST(2.0) * CUDA_PI)) * yr * (CST(3.0) * xr2 - yr2);
						break;
					case -2:
						Y = SQRT(CST(105.0) / (CST(4.0) * CUDA_PI)) * zr * yr * xr;
						break;
					case -1:
						Y = SQRT(CST(21.0) / (CST(16.0) * CST(2.0) * CUDA_PI)) * yr * (CST(4.0) * zr2 - xr2 - yr2);
						break;
					case 0:
						Y = SQRT(CST(7.0) / (CST(16.0) * CUDA_PI)) * zr
							* (CST(2.0) * zr2 - CST(3.0) * xr2 - CST(3.0) * yr2);
						break;
					case 1:
						Y = SQRT(CST(21.0) / (CST(16.0) * CST(2.0) * CUDA_PI)) * xr * (CST(4.0) * zr2 - xr2 - yr2);
						break;
					case 2:
						Y = SQRT(CST(105.0) / (CST(16.0) * CUDA_PI)) * zr * (xr2 - yr2);
						break;
					case 3:
						Y = SQRT(CST(35.0) / (CST(16.0) * CST(2.0) * CUDA_PI)) * xr * (xr2 - CST(3.0) * yr2);
						break;
				}
				break;
#endif	// L2 >= 3
#if L2 >= 4
			case 4:
				switch (m) {
					case -4:
						Y = SQRT((CST(35.0) * CST(9.0)) / (CST(16.0) * CUDA_PI)) * yr * xr * (xr2 - yr2);
						break;
					case -3:
						Y = SQRT((CST(9.0) * CST(35.0)) / (CST(16.0) * CST(2.0) * CUDA_PI)) * yr * zr
							* (CST(3.0) * xr2 - yr2);
						break;
					case -2:
						Y = SQRT((CST(9.0) * CST(5.0)) / (CST(16.0) * CUDA_PI)) * yr * xr
							* (CST(7.0) * zr2 - (xr2 + yr2 + zr2));
						break;
					case -1:
						Y = SQRT((CST(9.0) * CST(5.0)) / (CST(16.0) * CST(2.0) * CUDA_PI)) * yr * zr
							* (CST(7.0) * zr2 - CST(3.0) * (xr2 + yr2 + zr2));
						break;
					case 0:
						Y = SQRT(CST(9.0) / (CST(16.0) * CST(16.0) * CUDA_PI))
							* (CST(35.0) * zr2 * zr2 - CST(30.0) * zr2 + CST(3.0));
						break;
					case 1:
						Y = SQRT((CST(9.0) * CST(5.0)) / (CST(16.0) * CST(2.0) * CUDA_PI)) * xr * zr
							* (CST(7.0) * zr2 - CST(3.0) * (xr2 + yr2 + zr2));
						break;
					case 2:
						Y = SQRT((CST(9.0) * CST(5.0)) / (CST(8.0) * CST(8.0) * CUDA_PI)) * (xr2 - yr2)
							* (CST(7.0) * zr2 - (xr2 + yr2 + zr2));
						break;
					case 3:
						Y = SQRT((CST(9.0) * CST(35.0)) / (CST(16.0) * CST(2.0) * CUDA_PI)) * xr * zr
							* (xr2 - CST(3.0) * yr2);
						break;
					case 4:
						Y = SQRT((CST(9.0) * CST(35.0)) / (CST(16.0) * CST(16.0) * CUDA_PI))
							* (xr2 * (xr2 - CST(3.0) * yr2) - yr2 * (CST(3.0) * xr2 - yr2));
						break;
				}
				break;
#endif	// L2 >= 4
#if L2 >= 5
			case 5:
				switch (m) {
					case -5:
						Y = (CST(3.0) / CST(16.0)) * SQRT(CST(77.0) / (CST(2.0) * CUDA_PI)) * sint2 * sint2 * sint
							* SIN(CST(5.0) * phi);
						break;
					case -4:
						Y = (CST(3.0) / CST(8.0)) * SQRT(CST(385.0) / (CST(2.0) * CUDA_PI)) * sint2 * sint2
							* SIN(CST(4.0) * phi);
						break;
					case -3:
						Y = (CST(1.0) / CST(16.0)) * SQRT(CST(385.0) / (CST(2.0) * CUDA_PI)) * sint2 * sint
							* (CST(9.0) * cost2 - CST(1.0)) * SIN(CST(3.0) * phi);
						break;
					case -2:
						Y = (CST(1.0) / CST(4.0)) * SQRT(CST(1155.0) / (CST(4.0) * CUDA_PI)) * sint2
							* (CST(3.0) * cost2 * cost - cost) * SIN(CST(2.0) * phi);
						break;
					case -1:
						Y = (CST(1.0) / CST(8.0)) * SQRT(CST(165.0) / (CST(4.0) * CUDA_PI)) * sint
							* (CST(21.0) * cost2 * cost2 - CST(14.0) * cost2 + 1) * SIN(phi);
						break;
					case 0:
						Y = (CST(1.0) / CST(16.0)) * SQRT(CST(11.0) / CUDA_PI)
							* (CST(63.0) * cost2 * cost2 * cost - CST(70.0) * cost2 * cost + CST(15.0) * cost);
						break;
					case 1:
						Y = (CST(1.0) / CST(8.0)) * SQRT(CST(165.0) / (CST(4.0) * CUDA_PI)) * sint
							* (CST(21.0) * cost2 * cost2 - CST(14.0) * cost2 + 1) * COS(phi);
						break;
					case 2:
						Y = (CST(1.0) / CST(4.0)) * SQRT(CST(1155.0) / (CST(4.0) * CUDA_PI)) * sint2
							* (CST(3.0) * cost2 * cost - cost) * COS(CST(2.0) * phi);
						break;
					case 3:
						Y = (CST(1.0) / CST(16.0)) * SQRT(CST(385.0) / (CST(2.0) * CUDA_PI)) * sint2 * sint
							* (CST(9.0) * cost2 - CST(1.0)) * COS(CST(3.0) * phi);
						break;
					case 4:
						Y = (CST(3.0) / CST(8.0)) * SQRT(CST(385.0) / (CST(2.0) * CUDA_PI)) * sint2 * sint2
							* COS(CST(4.0) * phi);
						break;
					case 5:
						Y = (CST(3.0) / CST(16.0)) * SQRT(CST(77.0) / (CST(2.0) * CUDA_PI)) * sint2 * sint2 * sint
							* COS(CST(5.0) * phi);
						break;
				}
				break;
#endif	// L2 >= 5
		}

		return R * Y;
	}

	template<typename PrecisionType>
	__device__ PrecisionType atomicAddPrecision(PrecisionType *addr, PrecisionType val)
	{
		return atomicAdd(addr, val);
	}

	template<>
	__device__ double atomicAddPrecision(double *address, double val)
	{
		unsigned long long int *address_as_ull = (unsigned long long int *)address;
		unsigned long long int old = *address_as_ull, assumed;

		do {
			assumed = old;
			old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));

			// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
		} while (assumed != old);

		return __longlong_as_double(old);
	}

	template<typename PrecisionType>
	__device__ void splattingAtPos(PrecisionType pos_x,
								   PrecisionType pos_y,
								   PrecisionType weight,
								   MultidimArrayCuda<PrecisionType> &mP,
								   MultidimArrayCuda<PrecisionType> &mW)
	{
		int i = static_cast<int>(CUDA_ROUND(pos_y));
		int j = static_cast<int>(CUDA_ROUND(pos_x));
		if (!IS_OUTSIDE2D(mP, i, j)) {
			atomicAddPrecision(&A2D_ELEM(mP, i, j), weight);
			atomicAddPrecision(&A2D_ELEM(mW, i, j), CST(1.0));
		}
	}

	template<typename PrecisionType>
	__device__ size_t findCuda(const PrecisionType *begin, size_t size, PrecisionType value)
	{
		if (size <= 0) {
			return 0;
		}
		for (size_t i = 0; i < size; i++) {
			if (begin[i] == value) {
				return i;
			}
		}
		return size - 1;
	}

	template<typename PrecisionType>
	__device__ PrecisionType interpolatedElement2DCuda(PrecisionType x,
													   PrecisionType y,
													   const MultidimArrayCuda<PrecisionType> &diffImage)
	{
		int x0 = CUDA_FLOOR(x);
		PrecisionType fx = x - x0;
		int x1 = x0 + 1;
		int y0 = CUDA_FLOOR(y);
		PrecisionType fy = y - y0;
		int y1 = y0 + 1;

		int i0 = STARTINGY(diffImage);
		int j0 = STARTINGX(diffImage);
		int iF = FINISHINGY(diffImage);
		int jF = FINISHINGX(diffImage);

#define ASSIGNVAL2DCUDA(d, i, j)                      \
	if ((j) < j0 || (j) > jF || (i) < i0 || (i) > iF) \
		d = (PrecisionType)0;                         \
	else                                              \
		d = A2D_ELEM(diffImage, i, j);

		PrecisionType d00, d10, d11, d01;
		ASSIGNVAL2DCUDA(d00, y0, x0);
		ASSIGNVAL2DCUDA(d01, y0, x1);
		ASSIGNVAL2DCUDA(d10, y1, x0);
		ASSIGNVAL2DCUDA(d11, y1, x1);

		PrecisionType d0 = LIN_INTERP(fx, d00, d01);
		PrecisionType d1 = LIN_INTERP(fx, d10, d11);
		return LIN_INTERP(fy, d0, d1);
	}

}  // namespace device

/*
 * The first beast
 */
template<typename PrecisionType, bool usesZernike>
__global__ void forwardKernel(const MultidimArrayCuda<PrecisionType> cudaMV,
							  const MultidimArrayCuda<int> cudaVRecMaskF,
							  MultidimArrayCuda<PrecisionType> *cudaP,
							  MultidimArrayCuda<PrecisionType> *cudaW,
							  const int lastZ,
							  const int lastY,
							  const int lastX,
							  const int step,
							  const size_t sigma_size,
							  const PrecisionType *cudaSigma,
							  const PrecisionType iRmaxF,
							  const size_t idxY0,
							  const size_t idxZ0,
							  const int *cudaVL1,
							  const int *cudaVN,
							  const int *cudaVL2,
							  const int *cudaVM,
							  const PrecisionType *cudaClnm,
							  const PrecisionType *cudaR)
{
	int cubeX = threadIdx.x;
	int cubeY = threadIdx.y + blockIdx.y * blockDim.y;
	int cubeZ = blockIdx.z;
	int k = STARTINGZ(cudaMV) + cubeZ;
	int i = STARTINGY(cudaMV) + cubeY;
	int j = STARTINGX(cudaMV) + cubeX;
	if (cubeX % step != 0 || cubeY % step != 0 || cubeZ % step != 0) {
		return;
	}
	PrecisionType gx = 0.0, gy = 0.0, gz = 0.0;
	if (A3D_ELEM(cudaVRecMaskF, k, i, j) != 0) {
		int img_idx = 0;
		if (sigma_size > 1) {
			PrecisionType sigma_mask = A3D_ELEM(cudaVRecMaskF, k, i, j);
			img_idx = device::findCuda(cudaSigma, sigma_size, sigma_mask);
		}
		auto &mP = cudaP[img_idx];
		auto &mW = cudaW[img_idx];
		if (usesZernike) {
			auto k2 = k * k;
			auto kr = k * iRmaxF;
			auto k2i2 = k2 + i * i;
			auto ir = i * iRmaxF;
			auto r2 = k2i2 + j * j;
			auto jr = j * iRmaxF;
			auto rr = SQRT(r2) * iRmaxF;
			for (size_t idx = 0; idx < idxY0; idx++) {
				auto l1 = cudaVL1[idx];
				auto n = cudaVN[idx];
				auto l2 = cudaVL2[idx];
				auto m = cudaVM[idx];
				if (rr > 0 || l2 == 0) {
					PrecisionType zsph = device::ZernikeSphericalHarmonics(l1, n, l2, m, jr, ir, kr, rr);
					gx += cudaClnm[idx] * (zsph);
					gy += cudaClnm[idx + idxY0] * (zsph);
					gz += cudaClnm[idx + idxZ0] * (zsph);
				}
			}
		}

		auto r_x = j + gx;
		auto r_y = i + gy;
		auto r_z = k + gz;

		auto pos_x = cudaR[0] * r_x + cudaR[1] * r_y + cudaR[2] * r_z;
		auto pos_y = cudaR[3] * r_x + cudaR[4] * r_y + cudaR[5] * r_z;
		PrecisionType voxel_mV = A3D_ELEM(cudaMV, k, i, j);
		device::splattingAtPos(pos_x, pos_y, voxel_mV, mP, mW);
	}
}

/*
 * The second beast
 */
template<typename PrecisionType, bool usesZernike>
__global__ void backwardKernel(MultidimArrayCuda<PrecisionType> cudaMV,
							   const MultidimArrayCuda<PrecisionType> cudaMId,
							   const MultidimArrayCuda<int> VRecMaskB,
							   const int lastZ,
							   const int lastY,
							   const int lastX,
							   const int step,
							   const PrecisionType iRmaxF,
							   const size_t idxY0,
							   const size_t idxZ0,
							   const int *cudaVL1,
							   const int *cudaVN,
							   const int *cudaVL2,
							   const int *cudaVM,
							   const PrecisionType *cudaClnm,
							   const PrecisionType *cudaR)
{
	int cubeX = threadIdx.x;
	int cubeY = threadIdx.y + blockIdx.y * blockDim.y;
	int cubeZ = blockIdx.z;
	int k = STARTINGZ(cudaMV) + cubeZ;
	int i = STARTINGY(cudaMV) + cubeY;
	int j = STARTINGX(cudaMV) + cubeX;
	PrecisionType gx = 0.0, gy = 0.0, gz = 0.0;
	if (A3D_ELEM(VRecMaskB, k, i, j) != 0) {
		if (usesZernike) {
			auto k2 = k * k;
			auto kr = k * iRmaxF;
			auto k2i2 = k2 + i * i;
			auto ir = i * iRmaxF;
			auto r2 = k2i2 + j * j;
			auto jr = j * iRmaxF;
			auto rr = SQRT(r2) * iRmaxF;
			for (size_t idx = 0; idx < idxY0; idx++) {
				auto l1 = cudaVL1[idx];
				auto n = cudaVN[idx];
				auto l2 = cudaVL2[idx];
				auto m = cudaVM[idx];
				if (rr > 0 || l2 == 0) {
					PrecisionType zsph = device::ZernikeSphericalHarmonics(l1, n, l2, m, jr, ir, kr, rr);
					gx += cudaClnm[idx] * (zsph);
					gy += cudaClnm[idx + idxY0] * (zsph);
					gz += cudaClnm[idx + idxZ0] * (zsph);
				}
			}
		}

		auto r_x = j + gx;
		auto r_y = i + gy;
		auto r_z = k + gz;

		auto pos_x = cudaR[0] * r_x + cudaR[1] * r_y + cudaR[2] * r_z;
		auto pos_y = cudaR[3] * r_x + cudaR[4] * r_y + cudaR[5] * r_z;
		PrecisionType voxel = device::interpolatedElement2DCuda(pos_x, pos_y, cudaMId);
		A3D_ELEM(cudaMV, k, i, j) += voxel;
	}
}
}  // namespace cuda_forward_art_zernike3D
#endif	//CUDA_FORWARD_ART_ZERNIKE3D_CU
