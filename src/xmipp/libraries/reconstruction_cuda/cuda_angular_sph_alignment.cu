#include "hip/hip_runtime.h"
#pragma once

// Compilation settings

#ifndef KTT_USED
#include "cuda_volume_deform_sph_defines.h"
#include "cuda_angular_sph_alignment.h"
#endif

namespace AngularAlignmentGpu {
#if USE_DOUBLE_PRECISION == 1
// Types
using PrecisionType = double;
using PrecisionType3 = double3;
// Constants
#define _PI_ (3.1415926535897931e+0)
// Functions
#define SQRT sqrt
#define ATAN2 atan2
#define COS cos
#define SIN sin
#define CUDA_FLOOR floor

#if __CUDA_ARCH__ < 600
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

#else
// Types
using PrecisionType = float;
using PrecisionType3 = float3;
// Constants
#define _PI_ (3.1415926535897f)
// Functions
#define SQRT sqrtf
#define ATAN2 atan2f
#define COS cosf
#define SIN sinf
#define CUDA_FLOOR floorf

#endif// USE_DOUBLE_PRECISION

// Compilation settings - end

// Define data structures
#ifdef KTT_USED

struct ImageMetaData
{
    int xShift;
    int yShift;
    int zShift;

    int xDim;
    int yDim;
    int zDim;
}

#endif// KTT_USED

// CUDA kernel defines
#define BLOCK_SIZE (BLOCK_X_DIM * BLOCK_Y_DIM * BLOCK_Z_DIM)

// ImageData macros

// Index to global memory
#define GET_IDX(ImD,k,i,j) \
    ((ImD).xDim * (ImD).yDim * (k) + (ImD).xDim * (i) + (j))

// Logical index = Physical index + shift
#define P2L_X_IDX(ImD,j) \
    ((j) + (ImD).xShift)

#define P2L_Y_IDX(ImD,i) \
    ((i) + (ImD).yShift)

#define P2L_Z_IDX(ImD,k) \
    ((k) + (ImD).zShift)

// Physical index = Logical index - shift
#define L2P_X_IDX(ImD,j) \
    ((j) - (ImD).xShift)

#define L2P_Y_IDX(ImD,i) \
    ((i) - (ImD).yShift)

#define L2P_Z_IDX(ImD,k) \
    ((k) - (ImD).zShift)

// Element access
#define ELEM_3D(ImD,meta,k,i,j) \
    ((ImD)[GET_IDX((meta), (k), (i), (j))])

#define ELEM_2D(data,ImD,i,j) \
    ((data)[GET_IDX((ImD), 0, (i), (j))])

// TEST PRO ATOMIC_ADD
#define ELEM_2D_ADDR(data,ImD,i,j) \
    ((data) + GET_IDX((ImD), 0, (i), (j)))

#define ELEM_3D_SHIFTED(ImD,meta,k,i,j) \
    (ELEM_3D((ImD), (meta), (k) - (meta).zShift, (i) - (meta).yShift, (j) - (meta).xShift))

#define ELEM_2D_SHIFTED(data,ImD,i,j) \
    (ELEM_2D((data), (ImD), (i) - (ImD).yShift, (j) - (ImD).xShift))

// TEST PRO ATOMIC_ADD
#define ELEM_2D_SHIFTED_ADDR(data,ImD,i,j) \
    (ELEM_2D_ADDR((data), (ImD), (i) - (ImD).yShift, (j) - (ImD).xShift))

// Utility macros
#define IS_OUTSIDE(ImD,k,i,j) \
    ((j) < (ImD).xShift || (j) > (ImD).xShift + (ImD).xDim - 1 || \
     (i) < (ImD).yShift || (i) > (ImD).yShift + (ImD).yDim - 1 || \
     (k) < (ImD).zShift || (k) > (ImD).zShift + (ImD).zDim - 1)

#define IS_OUTSIDE_PHYS(ImD,k,i,j) \
    ((j) < 0 || (ImD).xDim <= (j) || \
     (i) < 0 || (ImD).yDim <= (i) || \
     (k) < 0 || (ImD).zDim <= (k))

// Smart casting to selected precision (at compile time)
// ...just shorter static_cast
#define CST(num) (static_cast<PrecisionType>((num)))

#define LIN_INTERP(a, l, h) ((l) + ((h) - (l)) * (a))

// For the current supported degrees L1, L2, the max is 56 coeficients
// if there is added support for higher degrees of L1, L2 then the
// max number of coeficient NEEDS to be recalculated and updated
#ifndef MAX_COEF_COUNT
#define MAX_COEF_COUNT 56
#endif

__constant__ PrecisionType3 cClnm[MAX_COEF_COUNT];
__constant__ int4 cZsh[MAX_COEF_COUNT];
__constant__ PrecisionType cRotation[3*3];

// Forward declarations
template<int _L1 = 5, int _L2 = 5>
__forceinline__ __device__ PrecisionType ZernikeSphericalHarmonics(int l1, int n, int l2, int m,
        PrecisionType xr, PrecisionType yr, PrecisionType zr, PrecisionType r);

__device__ PrecisionType interpolatedElement3D(
        const PrecisionType* ImD, ImageMetaData imgMeta,
        PrecisionType x, PrecisionType y, PrecisionType z,
        PrecisionType doutside_value = 0);

__forceinline__ __device__ void rotateCoordinates(PrecisionType* pos, const PrecisionType* rotation)
{
    PrecisionType tmp[3] = {0};

    for (size_t i = 0; i < 3; i++)
        for (size_t j = 0; j < 3; j++)
            tmp[i] += rotation[3 * i + j] * pos[j];

    pos[0] = tmp[0];
    pos[1] = tmp[1];
    pos[2] = tmp[2];
}

template<int _BLOCK_SIZE = BLOCK_SIZE, int _L1 = 5, int _L2 = 5>
__global__ void projectionKernel(
        PrecisionType Rmax2,
        PrecisionType iRmax,
        ImageMetaData volMeta,
        const PrecisionType* volData,
        const PrecisionType* rotation,
        unsigned steps,
        const int4* zshparams,
        const PrecisionType3* clnm,
        const int* volMask,
        PrecisionType* projectionPlane,
        PrecisionType* outArrayGlobal
        )
{
    extern __shared__ char sharedBuffer[];
    //unsigned sharedBufferOffset = 0;

    // Thread index in a block
    unsigned tIdx = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    // Get physical indexes
    int kPhys = blockIdx.z * blockDim.z + threadIdx.z;
    int iPhys = blockIdx.y * blockDim.y + threadIdx.y;
    int jPhys = blockIdx.x * blockDim.x + threadIdx.x;

    // Update to logical indexes (calculations expect logical indexing)
    PrecisionType pos[3];
    pos[2] = P2L_Z_IDX(volMeta, kPhys);
    pos[1] = P2L_Y_IDX(volMeta, iPhys);
    pos[0] = P2L_X_IDX(volMeta, jPhys);

    rotateCoordinates(pos, cRotation);
/*
    int4* zshShared = (int4*)(sharedBuffer + sharedBufferOffset);
    sharedBufferOffset += sizeof(int4) * steps;

    PrecisionType3* clnmShared = (PrecisionType3*)(sharedBuffer + sharedBufferOffset);
    sharedBufferOffset += sizeof(PrecisionType3) * steps;

    // Load zsh, clnm parameters to the shared memory
    if (steps <= _BLOCK_SIZE) {
        if (tIdx < steps) {
            zshShared[tIdx] = zshparams[tIdx];
            clnmShared[tIdx] = clnm[tIdx];
        }
    } else {
        if (tIdx == 0) {
            for (unsigned idx = 0; idx < steps; idx++) {
                zshShared[idx] = zshparams[idx];
                clnmShared[idx] = clnm[idx];
            }
        }
    }

    __syncthreads();
*/
    // Define and compute necessary values
    PrecisionType r2 = pos[2]*pos[2] + pos[1]*pos[1] + pos[0]*pos[0];
    PrecisionType rr = SQRT(r2) * iRmax;
    PrecisionType gx = 0.0, gy = 0.0, gz = 0.0;

    if (r2 < Rmax2) {
        for (unsigned idx = 0; idx < steps; idx++) {
            int l1 = cZsh[idx].w;
            int n = cZsh[idx].x;
            int l2 = cZsh[idx].y;
            int m = cZsh[idx].z;

            PrecisionType zsph = ZernikeSphericalHarmonics<_L1, _L2>(l1, n, l2, m,
                    pos[0] * iRmax, pos[1] * iRmax, pos[2] * iRmax, rr);

            if (rr > 0 || l2 == 0) {
                gx += zsph * cClnm[idx].x;
                gy += zsph * cClnm[idx].y;
                gz += zsph * cClnm[idx].z;
            }
        }
    }

    int maskVoxel;
    int kMask = (int)(pos[2] + gz);
    int iMask = (int)(pos[1] + gy);
    int jMask = (int)(pos[0] + gx);

    if (IS_OUTSIDE(volMeta, kMask, iMask, jMask)) {
        maskVoxel = 0;
    } else {
        maskVoxel = ELEM_3D_SHIFTED(volMask, volMeta, kMask, iMask, jMask);
    }

    PrecisionType localCount = 0.0, localSumVD = 0.0, localModg = 0.0;

    if (maskVoxel == 1) {
        PrecisionType voxelI = interpolatedElement3D(volData, volMeta,
                pos[0] + gx, pos[1] + gy, pos[2] + gz);
        //ELEM_2D_SHIFTED(projectionPlane, volMeta,
        //        P2L_Y_IDX(volMeta, iPhys), P2L_X_IDX(volMeta, jPhys)) += voxelI;
        atomicAdd(ELEM_2D_SHIFTED_ADDR(projectionPlane, volMeta,
                P2L_Y_IDX(volMeta, iPhys), P2L_X_IDX(volMeta, jPhys)), voxelI);
        localSumVD += voxelI;
        localModg += gx*gx + gy*gy + gz*gz;
        localCount++;
    }

    __shared__ PrecisionType countShared[_BLOCK_SIZE];
    __shared__ PrecisionType sumVDShared[_BLOCK_SIZE];
    __shared__ PrecisionType modfgShared[_BLOCK_SIZE];

    countShared[tIdx] = localCount;
    sumVDShared[tIdx] = localSumVD;
    modfgShared[tIdx] = localModg;

    __syncthreads();

    // First level of conditions are evaluated during compilation
    if (_BLOCK_SIZE >= 1024) {
        if (tIdx < 512) {
            countShared[tIdx] += countShared[tIdx + 512];
            sumVDShared[tIdx] += sumVDShared[tIdx + 512];
            modfgShared[tIdx] += modfgShared[tIdx + 512];
        }
        __syncthreads();
    }
    if (_BLOCK_SIZE >= 512) {
        if (tIdx < 256) {
            countShared[tIdx] += countShared[tIdx + 256];
            sumVDShared[tIdx] += sumVDShared[tIdx + 256];
            modfgShared[tIdx] += modfgShared[tIdx + 256];
        }
        __syncthreads();
    }
    if (_BLOCK_SIZE >= 256) {
        if (tIdx < 128) {
            countShared[tIdx] += countShared[tIdx + 128];
            sumVDShared[tIdx] += sumVDShared[tIdx + 128];
            modfgShared[tIdx] += modfgShared[tIdx + 128];
        }
        __syncthreads();
    }
    if (_BLOCK_SIZE >= 128) {
        if (tIdx < 64) {
            countShared[tIdx] += countShared[tIdx + 64];
            sumVDShared[tIdx] += sumVDShared[tIdx + 64];
            modfgShared[tIdx] += modfgShared[tIdx + 64];
        }
        __syncthreads();
    }
    // Last warp reduction
    if (tIdx < 32) {
        localCount = countShared[tIdx];
        localSumVD = sumVDShared[tIdx];
        localModg = modfgShared[tIdx];
        if (_BLOCK_SIZE >= 64) {
            localCount += countShared[tIdx + 32];
            localSumVD += sumVDShared[tIdx + 32];
            localModg += modfgShared[tIdx + 32];
        }
        // Reduce warp
        for (int offset = 32 / 2; offset > 0; offset >>= 1) {
            localCount += __shfl_down_sync(0xFFFFFFFF, localCount, offset);
            localSumVD += __shfl_down_sync(0xFFFFFFFF, localSumVD, offset);
            localModg += __shfl_down_sync(0xFFFFFFFF, localModg, offset);
        }
    }

    // Save values to the global memory for later
    if (tIdx == 0) {
        unsigned bIdx = blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x;
        unsigned GRID_SIZE = gridDim.x * gridDim.y * gridDim.z;
        // Resulting values are in variables local* => no need to go into shared mem
        outArrayGlobal[bIdx] = localCount;
        outArrayGlobal[bIdx + GRID_SIZE] = localSumVD;
        outArrayGlobal[bIdx + GRID_SIZE * 2] = localModg;
    }
}

/*
 * Linear interpolation
 */
__device__ PrecisionType interpolatedElement3D(
        const PrecisionType* ImD, ImageMetaData imgMeta,
        PrecisionType x, PrecisionType y, PrecisionType z,
        PrecisionType outside_value)
{
        int x0 = (int)CUDA_FLOOR(x);
        PrecisionType fx = x - x0;
        int x1 = x0 + 1;

        int y0 = (int)CUDA_FLOOR(y);
        PrecisionType fy = y - y0;
        int y1 = y0 + 1;

        int z0 = (int)CUDA_FLOOR(z);
        PrecisionType fz = z - z0;
        int z1 = z0 + 1;

        PrecisionType d000 = (IS_OUTSIDE(imgMeta, z0, y0, x0)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, imgMeta, z0, y0, x0);
        PrecisionType d001 = (IS_OUTSIDE(imgMeta, z0, y0, x1)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, imgMeta, z0, y0, x1);
        PrecisionType d010 = (IS_OUTSIDE(imgMeta, z0, y1, x0)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, imgMeta, z0, y1, x0);
        PrecisionType d011 = (IS_OUTSIDE(imgMeta, z0, y1, x1)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, imgMeta, z0, y1, x1);
        PrecisionType d100 = (IS_OUTSIDE(imgMeta, z1, y0, x0)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, imgMeta, z1, y0, x0);
        PrecisionType d101 = (IS_OUTSIDE(imgMeta, z1, y0, x1)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, imgMeta, z1, y0, x1);
        PrecisionType d110 = (IS_OUTSIDE(imgMeta, z1, y1, x0)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, imgMeta, z1, y1, x0);
        PrecisionType d111 = (IS_OUTSIDE(imgMeta, z1, y1, x1)) ?
            outside_value : ELEM_3D_SHIFTED(ImD, imgMeta, z1, y1, x1);

        PrecisionType dx00 = LIN_INTERP(fx, d000, d001);
        PrecisionType dx01 = LIN_INTERP(fx, d100, d101);
        PrecisionType dx10 = LIN_INTERP(fx, d010, d011);
        PrecisionType dx11 = LIN_INTERP(fx, d110, d111);
        PrecisionType dxy0 = LIN_INTERP(fy, dx00, dx10);
        PrecisionType dxy1 = LIN_INTERP(fy, dx01, dx11);

        return LIN_INTERP(fz, dxy0, dxy1);
}

template<int _L1, int _L2>
__forceinline__ __device__ PrecisionType ZernikeSphericalHarmonics(int l1, int n, int l2, int m,
        PrecisionType xr, PrecisionType yr, PrecisionType zr, PrecisionType rr)
{
    // General variables
    PrecisionType r2 = rr * rr, xr2 = xr * xr, yr2 = yr * yr,
                  zr2 = zr * zr;


    // Variables needed for l2 >= 5
    PrecisionType tht = CST(0.0), phi = CST(0.0), cost = CST(0.0),
                  sint = CST(0.0), cost2 = CST(0.0), sint2 = CST(0.0);
    if (_L2 >= 5) {
        if (l2 >= 5) {
            tht = ATAN2(yr, xr);
            phi = ATAN2(zr, SQRT(xr2 + yr2));
            sint = SIN(phi);
            cost = COS(tht);
            sint2 = sint * sint;
            cost2 = cost * cost;
        }
    }

    // Zernike polynomial
    PrecisionType R = CST(0.0);

    switch (l1) {
        case 0:
            R = SQRT(CST(3));
            break;
        case 1:
            R = SQRT(CST(5)) * rr;
            break;
        case 2:
            switch (n) {
                case 0:
                    R = CST(-0.5) * SQRT(CST(7)) *
                        (CST(2.5) * (1 - 2 * r2) + CST(0.5));
                    break;
                case 2:
                    R = SQRT(CST(7)) * r2;
                    break;
            }
            break;
            if (_L1 >= 3) {
                case 3:
                    switch (n) {
                        case 1:
                            R = CST(-1.5) * rr * (CST(3.5) * (1 - 2 * r2) + CST(1.5));
                            break;
                        case 3:
                            R = 3 * r2 * rr;
                    }
                    break;
            }
            if (_L1 >= 4) {
                case 4:
                    switch (n) {
                        case 0:
                            R = SQRT(CST(11)) *
                                ((63 * r2 * r2 / 8) - (35 * r2 / 4) + (CST(15) / CST(8)));
                            break;
                        case 2:
                            R = CST(-0.5) * SQRT(CST(11)) * r2 *
                                (CST(4.5) * (1 - 2 * r2) + CST(2.5));
                            break;
                        case 4:
                            R = SQRT(CST(11)) * r2 * r2;
                            break;
                    }
                    break;
            }
            if (_L1 >= 5) {
                case 5:
                    switch (n) {
                        case 1:
                            R = SQRT(CST(13)) * rr *
                                ((99 * r2 * r2 / 8) - (63 * r2 / 4) + (CST(35) / CST(8)));
                            break;
                        case 3:
                            R = CST(-0.5) * SQRT(CST(13)) * r2 * rr *
                                (CST(5.5) * (1 - 2 * r2) + CST(3.5));
                            break;
                    }
                    break;
            }
    }

    // Spherical harmonic
    PrecisionType Y = CST(0.0);

    switch (l2) {
        case 0:
            Y = (CST(1.0) / CST(2.0)) * SQRT((PrecisionType)CST(1.0) / _PI_);
            break;
        case 1:
            switch (m) {
                case -1:
                    Y = SQRT(CST(3.0) / (CST(4.0) * _PI_)) * yr;
                    break;
                case 0:
                    Y = SQRT(CST(3.0) / (CST(4.0) * _PI_)) * zr;
                    break;
                case 1:
                    Y = SQRT(CST(3.0) / (CST(4.0) * _PI_)) * xr;
                    break;
            }
            break;
        case 2:
            switch (m) {
                case -2:
                    Y = SQRT(CST(15.0) / (CST(4.0) * _PI_)) * xr * yr;
                    break;
                case -1:
                    Y = SQRT(CST(15.0) / (CST(4.0) * _PI_)) * zr * yr;
                    break;
                case 0:
                    Y = SQRT(CST(5.0) / (CST(16.0) * _PI_)) *
                        (-xr2 - yr2 + CST(2.0) * zr2);
                    break;
                case 1:
                    Y = SQRT(CST(15.0) / (CST(4.0) * _PI_)) * xr * zr;
                    break;
                case 2:
                    Y = SQRT(CST(15.0) / (CST(16.0) * _PI_)) * (xr2 - yr2);
                    break;
            }
            break;
            if (_L2 >= 3) {
                case 3:
                    switch (m) {
                        case -3:
                            Y = SQRT(CST(35.0) / (CST(16.0) * CST(2.0) * _PI_)) * yr *
                                (CST(3.0) * xr2 - yr2);
                            break;
                        case -2:
                            Y = SQRT(CST(105.0) / (CST(4.0) * _PI_)) * zr * yr * xr;
                            break;
                        case -1:
                            Y = SQRT(CST(21.0) / (CST(16.0) * CST(2.0) * _PI_)) * yr *
                                (CST(4.0) * zr2 - xr2 - yr2);
                            break;
                        case 0:
                            Y = SQRT(CST(7.0) / (CST(16.0) * _PI_)) * zr *
                                (CST(2.0) * zr2 - CST(3.0) * xr2 - CST(3.0) * yr2);
                            break;
                        case 1:
                            Y = SQRT(CST(21.0) / (CST(16.0) * CST(2.0) * _PI_)) * xr *
                                (CST(4.0) * zr2 - xr2 - yr2);
                            break;
                        case 2:
                            Y = SQRT(CST(105.0) / (CST(16.0) * _PI_)) * zr * (xr2 - yr2);
                            break;
                        case 3:
                            Y = SQRT(CST(35.0) / (CST(16.0) * CST(2.0) * _PI_)) * xr *
                                (xr2 - CST(3.0) * yr2);
                            break;
                    }
                    break;
            }
            if (_L2 >= 4) {
                case 4:
                    switch (m) {
                        case -4:
                            Y = SQRT((CST(35.0) * CST(9.0)) / (CST(16.0) * _PI_)) * yr *
                                xr * (xr2 - yr2);
                            break;
                        case -3:
                            Y = SQRT((CST(9.0) * CST(35.0)) /
                                    (CST(16.0) * CST(2.0) * _PI_)) *
                                yr * zr * (CST(3.0) * xr2 - yr2);
                            break;
                        case -2:
                            Y = SQRT((CST(9.0) * CST(5.0)) / (CST(16.0) * _PI_)) * yr * xr *
                                (CST(7.0) * zr2 - (xr2 + yr2 + zr2));
                            break;
                        case -1:
                            Y = SQRT((CST(9.0) * CST(5.0)) /
                                    (CST(16.0) * CST(2.0) * _PI_)) *
                                yr * zr * (CST(7.0) * zr2 - CST(3.0) * (xr2 + yr2 + zr2));
                            break;
                        case 0:
                            Y = SQRT(CST(9.0) / (CST(16.0) * CST(16.0) * _PI_)) *
                                (CST(35.0) * zr2 * zr2 - CST(30.0) * zr2 + CST(3.0));
                            break;
                        case 1:
                            Y = SQRT((CST(9.0) * CST(5.0)) /
                                    (CST(16.0) * CST(2.0) * _PI_)) *
                                xr * zr * (CST(7.0) * zr2 - CST(3.0) * (xr2 + yr2 + zr2));
                            break;
                        case 2:
                            Y = SQRT((CST(9.0) * CST(5.0)) / (CST(8.0) * CST(8.0) * _PI_)) *
                                (xr2 - yr2) * (CST(7.0) * zr2 - (xr2 + yr2 + zr2));
                            break;
                        case 3:
                            Y = SQRT((CST(9.0) * CST(35.0)) /
                                    (CST(16.0) * CST(2.0) * _PI_)) *
                                xr * zr * (xr2 - CST(3.0) * yr2);
                            break;
                        case 4:
                            Y = SQRT((CST(9.0) * CST(35.0)) /
                                    (CST(16.0) * CST(16.0) * _PI_)) *
                                (xr2 * (xr2 - CST(3.0) * yr2) -
                                 yr2 * (CST(3.0) * xr2 - yr2));
                            break;
                    }
                    break;
            }
            if (_L2 >= 5) {
                case 5:
                    switch (m) {
                        case -5:
                            Y = (CST(3.0) / CST(16.0)) *
                                SQRT(CST(77.0) / (CST(2.0) * _PI_)) * sint2 * sint2 * sint *
                                SIN(CST(5.0) * phi);
                            break;
                        case -4:
                            Y = (CST(3.0) / CST(8.0)) *
                                SQRT(CST(385.0) / (CST(2.0) * _PI_)) * sint2 * sint2 *
                                SIN(CST(4.0) * phi);
                            break;
                        case -3:
                            Y = (CST(1.0) / CST(16.0)) *
                                SQRT(CST(385.0) / (CST(2.0) * _PI_)) * sint2 * sint *
                                (CST(9.0) * cost2 - CST(1.0)) * SIN(CST(3.0) * phi);
                            break;
                        case -2:
                            Y = (CST(1.0) / CST(4.0)) *
                                SQRT(CST(1155.0) / (CST(4.0) * _PI_)) * sint2 *
                                (CST(3.0) * cost2 * cost - cost) * SIN(CST(2.0) * phi);
                            break;
                        case -1:
                            Y = (CST(1.0) / CST(8.0)) *
                                SQRT(CST(165.0) / (CST(4.0) * _PI_)) * sint *
                                (CST(21.0) * cost2 * cost2 - CST(14.0) * cost2 + 1) *
                                SIN(phi);
                            break;
                        case 0:
                            Y = (CST(1.0) / CST(16.0)) * SQRT(CST(11.0) / _PI_) *
                                (CST(63.0) * cost2 * cost2 * cost -
                                 CST(70.0) * cost2 * cost + CST(15.0) * cost);
                            break;
                        case 1:
                            Y = (CST(1.0) / CST(8.0)) *
                                SQRT(CST(165.0) / (CST(4.0) * _PI_)) * sint *
                                (CST(21.0) * cost2 * cost2 - CST(14.0) * cost2 + 1) *
                                COS(phi);
                            break;
                        case 2:
                            Y = (CST(1.0) / CST(4.0)) *
                                SQRT(CST(1155.0) / (CST(4.0) * _PI_)) * sint2 *
                                (CST(3.0) * cost2 * cost - cost) * COS(CST(2.0) * phi);
                            break;
                        case 3:
                            Y = (CST(1.0) / CST(16.0)) *
                                SQRT(CST(385.0) / (CST(2.0) * _PI_)) * sint2 * sint *
                                (CST(9.0) * cost2 - CST(1.0)) * COS(CST(3.0) * phi);
                            break;
                        case 4:
                            Y = (CST(3.0) / CST(8.0)) *
                                SQRT(CST(385.0) / (CST(2.0) * _PI_)) * sint2 * sint2 *
                                COS(CST(4.0) * phi);
                            break;
                        case 5:
                            Y = (CST(3.0) / CST(16.0)) *
                                SQRT(CST(77.0) / (CST(2.0) * _PI_)) * sint2 * sint2 * sint *
                                COS(CST(5.0) * phi);
                            break;
                    }
                    break;
            }
    }

    return R * Y;
}

} // namespace AngularAlignmentGpu
